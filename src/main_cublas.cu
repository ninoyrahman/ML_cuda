#include "hip/hip_runtime.h"
// main.cpp : Defines the entry point for the console application.
//
#include <iostream>
#include <math.h>
#include "util.h"
#include "kernel.h"
#include "hipblas.h"
 
// main routine that executes on the host
int main(void){

  // cublas handle
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // scaling factor
  float alpha = 1.0f;
  float beta  = 0.0f;  

  float *matA_h, *matB_h, *matC_h;  // Pointer to host & device arrays
  float *matA_d, *matB_d, *matC_d;  // Pointer to host & device arrays
  const int N1 = 4096; // matC row/matA row number
  const int N2 = 2048; // matC column/matB column number
  const int N3 = 1024;    // matB row/matA column number
  const int nthreads = 32; // number of threads

  size_t size_matA = N1 * N3 * sizeof(float);
  size_t size_matB = N3 * N2 * sizeof(float);
  size_t size_matC = N1 * N2 * sizeof(float);

  matA_h  = new float[N1 * N3];    // Allocate array on host
  matB_h  = new float[N3 * N2];    // Allocate array on host
  matC_h  = new float[N1 * N2];    // Allocate array on host

  float *matCr_h  = new float[N1 * N2];    // Allocate array on host
  
  hipMalloc((void **) &matA_d, size_matA);   // Allocate array on device
  hipMalloc((void **) &matB_d, size_matB);   // Allocate array on device
  hipMalloc((void **) &matC_d, size_matC);   // Allocate array on device

  // Initialize host array and copy it to CUDA device
  random_matrix(matA_h, N1, N3);
  random_matrix(matB_h, N3, N2); 
  
  // copy data from host to device
  float et;
  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);
  hipEventRecord(start);
  hipMemcpy(matA_d, matA_h, size_matA, hipMemcpyHostToDevice); // copy matA to device
  hipMemcpy(matB_d, matB_h, size_matB, hipMemcpyHostToDevice); // copy matB to device
  hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&et, start, stop);
  std::cout << "time elapsed(copy to device) = " << et << std::endl;

  // Do calculation on device
  dim3 threadsPerBlock(nthreads, nthreads);
  dim3 blocksPerGrid((int)ceil(N1 / threadsPerBlock.x), (int)ceil(N2 / threadsPerBlock.y));

  std::cout << " A matrix dims = (" << N1 << ", " << N3 << ")" << std::endl;
  std::cout << " B matrix dims = (" << N3 << ", " << N2 << ")" << std::endl;
  std::cout << " C matrix dims = (" << N1 << ", " << N2 << ")" << std::endl;     
  std::cout << " thread per block = (" << threadsPerBlock.x << ", " << threadsPerBlock.y << ")" << std::endl;
  std::cout << " block number = (" << blocksPerGrid.x << ", " << blocksPerGrid.y << ")" << std::endl;
  
  // gpu matrix multiplication
  hipEventRecord(start);
  status = hipblasSgemm(handle,
    HIPBLAS_OP_T, HIPBLAS_OP_T,
    N1, N2, N3,
    &alpha,
    matA_d, N3,
    matB_d, N2,
    &beta,
    matC_d, N1);
  hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&et, start, stop);
  std::cout << "time elapsed(kernel) = " << et << std::endl;

  if (status != HIPBLAS_STATUS_SUCCESS) 
    printf("hipblasSgemm returned error code %d\n", status);

  // Retrieve result from device and store it in host array
  hipEventRecord(start);
  hipMemcpy(matCr_h, matC_d, size_matC, hipMemcpyDeviceToHost);
  hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&et, start, stop);
  std::cout << "time elapsed(copy from device) = " << et << std::endl;

  // cpu matrix multiplication
  rearrange_matrix(matCr_h, matC_h, N1, N2);
  mat_mul_varify(matA_h, matB_h, matC_h, N1, N2, N3);
  
  // Print results
  // print_matrix(matC_h, N1, N2);
  
  // Cleanup
  delete [] matA_h;
  delete [] matB_h;
  delete [] matC_h;
  delete [] matCr_h;
  hipFree(matA_d);
  hipFree(matB_d);
  hipFree(matC_d);

  return 0;
}