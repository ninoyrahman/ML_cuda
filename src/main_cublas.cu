// main.cpp : Defines the entry point for the console application.
//
#include <iostream>
#include <math.h>
#include "util.h"
#include "kernel.h"
#include "hipblas.h"
 
// main routine that executes on the host
int main(void){

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////declaration///////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  // varify gpu calculation
  bool isVarify = true;

  // profile variable
  float et;
  hipEvent_t start, stop;

  // cublas handle
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // scaling factor
  float alpha;
  float beta;

  // matrix and vector size
  const int N1 = 4096; // matC row/matA row number
  const int N2 = 2048; // matC column/matB column number
  const int N3 = 1024; // matB row/matA column number

  // allocate on host
  float *matA_h  = new float[N1 * N3];
  float *matB_h  = new float[N3 * N2];
  float *matC_h  = new float[N1 * N2];
  float *vecB_h  = new float[N1];
  float *vecO_h  = new float[N2];
  
  // Pointer to device arrays
  float *matA_d, *matB_d, *matC_d;
  float *vecB_d, *vecO_d;

  // size of matrix
  size_t size_matA = N1 * N3 * sizeof(float);
  size_t size_matB = N3 * N2 * sizeof(float);
  size_t size_matC = N1 * N2 * sizeof(float);

  // size of vector
  size_t size_vecB = N1 * sizeof(float);
  size_t size_vecO = N2 * sizeof(float);

  // allocate on device
  hipMalloc((void **) &matA_d, size_matA);
  hipMalloc((void **) &matB_d, size_matB);
  hipMalloc((void **) &matC_d, size_matC);
  hipMalloc((void **) &vecB_d, size_vecB);
  hipMalloc((void **) &vecO_d, size_vecO);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////initialization////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////

  // initialize performance profile variables
  hipEventCreate(&start); hipEventCreate(&stop);

  // print matrix size
  std::cout << " A matrix dims = (" << N1 << ", " << N3 << ")" << std::endl;
  std::cout << " B matrix dims = (" << N3 << ", " << N2 << ")" << std::endl;
  std::cout << " C matrix dims = (" << N1 << ", " << N2 << ")" << std::endl;
  std::cout << " B vector dims = (" << N1 << ")" << std::endl;
  std::cout << " O vector dims = (" << N2 << ")" << std::endl;

  // Initialize host array and copy it to CUDA device
  random_matrix(matA_h, N1, N3);
  random_matrix(matB_h, N3, N2);
  random_vector(vecB_h, N1);
  setvalue_vector(vecO_h, N2, 1.0f);
  
  // copy data from host to device
  hipEventRecord(start);
  hipMemcpy(matA_d, matA_h, size_matA, hipMemcpyHostToDevice); // copy matA to device
  hipMemcpy(matB_d, matB_h, size_matB, hipMemcpyHostToDevice); // copy matB to device
  hipMemcpy(vecB_d, vecB_h, size_vecB, hipMemcpyHostToDevice); // copy vecB to device
  hipMemcpy(vecO_d, vecO_h, size_vecO, hipMemcpyHostToDevice); // copy vecO to device
  hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&et, start, stop);
  std::cout << "time elapsed(copy to device) = " << et << std::endl;

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////computation///////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////  
  
  // gpu matrix multiplication
  hipEventRecord(start);
  alpha = 1.0f;
  beta = 0.0f;
  status = hipblasSgemm(handle,
    HIPBLAS_OP_N, HIPBLAS_OP_N,
    N1, N2, N3,
    &alpha,
    matA_d, N1,
    matB_d, N3,
    &beta,
    matC_d, N1);
  hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&et, start, stop);
  std::cout << "time elapsed(mat x mat) = " << et << std::endl;

  if (status != HIPBLAS_STATUS_SUCCESS) 
    printf("hipblasSgemm returned error code %d\n", status);

  if (isVarify){
    // Retrieve result from device and store it in host array
    hipEventRecord(start);
    hipMemcpy(matC_h, matC_d, size_matC, hipMemcpyDeviceToHost);
    hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&et, start, stop);
    std::cout << "time elapsed(copy from device) = " << et << std::endl;

    // cpu matrix multiplication 
    // copy matC from device to host before calling varify routine
    mat_mul_varify(matA_h, matB_h, matC_h, N1, N2, N3);
  }

  // gpu matrix vector addition
  hipEventRecord(start);
  alpha = 1.0f;
  status = hipblasSger(handle, 
    N1, N2,
    &alpha,
    vecB_d, 1,
    vecO_d, 1,
    matC_d, N1);
  hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&et, start, stop);
  std::cout << "time elapsed(mat + vec) = " << et << std::endl;

  if (status != HIPBLAS_STATUS_SUCCESS) 
    printf("hipblasSger returned error code %d\n", status);

  // Retrieve result from device and store it in host array
  hipEventRecord(start);
  hipMemcpy(matC_h, matC_d, size_matC, hipMemcpyDeviceToHost);
  hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&et, start, stop);
  std::cout << "time elapsed(copy from device) = " << et << std::endl;    

  if (isVarify){
    // cpu matrix multiplication and vector addition 
    // copy matC from device to host before calling varify routine
    mat_mul_vec_sum_varify(matA_h, matB_h, matC_h, vecB_h, N1, N2, N3);
  }
  
  // Print results
  // print_matrix(matC_h, N1, N2);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////finalization//////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////    
  
  // Cleanup
  delete [] matA_h;
  delete [] matB_h;
  delete [] matC_h;
  delete [] vecB_h;
  delete [] vecO_h;
  hipFree(matA_d);
  hipFree(matB_d);
  hipFree(matC_d);
  hipFree(vecB_d);
  hipFree(vecO_d);

  return 0;
}