// main.cpp : Defines the entry point for the console application.
//
#include <iostream>
#include <math.h>
#include "util.h"
#include "kernal.h"
 
// main routine that executes on the host
int main(void){

  float *matA_h, *matB_h, *matC_h;  // Pointer to host & device arrays
  float *matA_d, *matB_d, *matC_d;  // Pointer to host & device arrays
  const int N1 = 8; // matC row/matA row number
  const int N2 = 8; // matC column/matB column number
  const int N3 = 8; // matB row/matA column number
  size_t size_matA = N1 * N3 * sizeof(float);
  size_t size_matB = N3 * N2 * sizeof(float);
  size_t size_matC = N1 * N2 * sizeof(float);

  matA_h = new float[N1 * N3];    // Allocate array on host
  matB_h = new float[N3 * N2];    // Allocate array on host
  matC_h = new float[N1 * N2];    // Allocate array on host
  hipMalloc((void **) &matA_d, size_matA);   // Allocate array on device
  hipMalloc((void **) &matB_d, size_matB);   // Allocate array on device
  hipMalloc((void **) &matC_d, size_matC);   // Allocate array on device

  // Initialize host array and copy it to CUDA device
  random_matrix(matA_h, N1, N3);
  random_matrix(matB_h, N3, N2);

  // copy data from host to device
  hipMemcpy(matA_d, matA_h, size_matA, hipMemcpyHostToDevice);
  hipMemcpy(matB_d, matB_h, size_matB, hipMemcpyHostToDevice);
  
  // Do calculation on device
  dim3 threadsPerBlock(4, 4);
  dim3 blocksPerGrid(N1 / threadsPerBlock.x, N2 / threadsPerBlock.y);

  std::cout << " thread per block = (" << threadsPerBlock.x << ", " << threadsPerBlock.y << ")" << std::endl;
  std::cout << " block number = (" << blocksPerGrid.x << ", " << blocksPerGrid.y << ")" << std::endl;
  
  // matrix multiplication
  mat_mul <<< blocksPerGrid, threadsPerBlock >>> (matA_d, matB_d, matC_d, N1, N2, N3);

  // Retrieve result from device and store it in host array
  hipMemcpy(matC_h, matC_d, size_matC, hipMemcpyDeviceToHost);
  
  // Print results
  print_matrix(matC_h, N1, N2);
  
  // Cleanup
  delete [] matA_h;
  delete [] matB_h;
  delete [] matC_h;
  hipFree(matA_d);
  hipFree(matB_d);
  hipFree(matC_d);

  return 0;
}