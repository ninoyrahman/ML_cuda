// main.cpp : Defines the entry point for the console application.
//
#include <iostream>
#include <math.h>
#include "hipblas.h"
#include "util.h"
#include "nn.h"

// main routine that executes on the host
int main(void){

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  /////////////////////////declaration and allocation///////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////

  // matrix and vector size
  const int Ns = 50; // matB row/matA column number
  const int N0 = 784;  // matC row/matA row number
  const int N1 = 256;   // matC column/matB column number
  const int N2 = 128;   // matB row/matA column number
  const int N3 = 10;   // matB row/matA column number

  // allocate on host

  // data X, Y
  float *matX_h  = new float[N0 * Ns];
  float *matY_h  = new float[N3 * Ns];

  // weights
  float *matw1_h  = new float[N1 * N0];
  float *matw2_h  = new float[N2 * N1];
  float *matw3_h  = new float[N3 * N2];
  
  // biases
  float *vecb1_h  = new float[N1];
  float *vecb2_h  = new float[N2];
  float *vecb3_h  = new float[N3];
  
  // Pointer to device arrays
  float *matX_d, *matY_d;
  float *matw1_d, *matw2_d, *matw3_d;
  float *vecb1_d, *vecb2_d, *vecb3_d;

  // size of matrix
  size_t size_matX  = N0 * Ns * sizeof(float);
  size_t size_matY  = N3 * Ns * sizeof(float);

  size_t size_matw1 = N1 * N0 * sizeof(float);
  size_t size_matw2 = N2 * N1 * sizeof(float);
  size_t size_matw3 = N3 * N2 * sizeof(float);

  // size of vector
  size_t size_vecb1 = N1 * sizeof(float);
  size_t size_vecb2 = N2 * sizeof(float);
  size_t size_vecb3 = N3 * sizeof(float);

  // allocate on device
  hipMalloc((void **) &matX_d, size_matX);
  hipMalloc((void **) &matY_d, size_matY);

  hipMalloc((void **) &matw1_d, size_matw1);
  hipMalloc((void **) &matw2_d, size_matw2);
  hipMalloc((void **) &matw3_d, size_matw3);

  hipMalloc((void **) &vecb1_d, size_vecb1);
  hipMalloc((void **) &vecb2_d, size_vecb2);
  hipMalloc((void **) &vecb3_d, size_vecb3);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////initialization////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////

  // print matrix size
  std::cout << " X matrix dims = (" << N0 << ", " << Ns << ")" << std::endl;
  std::cout << " Y matrix dims = (" << N0 << ", " << Ns << ")" << std::endl;

  std::cout << " w1 matrix dims = (" << N1 << ", " << N0 << ")" << std::endl;
  std::cout << " w2 matrix dims = (" << N2 << ", " << N1 << ")" << std::endl;
  std::cout << " w3 matrix dims = (" << N3 << ", " << N2 << ")" << std::endl;

  std::cout << " b1 vector dims = (" << N1 << ")" << std::endl;
  std::cout << " b2 vector dims = (" << N2 << ")" << std::endl;
  std::cout << " b3 vector dims = (" << N3 << ")" << std::endl;

  // Initialize host array and copy it to CUDA device
  random_matrix(matX_h, N0, Ns);
  random_matrix(matY_h, N3, Ns);

  random_matrix(matw1_h, N1, N0);
  random_matrix(matw2_h, N2, N1);
  random_matrix(matw3_h, N3, N2);

  random_vector(vecb1_h, N1);
  random_vector(vecb2_h, N2);
  random_vector(vecb3_h, N3);
  
  // copy data from host to device
  hipMemcpy(matX_d, matX_h, size_matX, hipMemcpyHostToDevice);
  hipMemcpy(matY_d, matY_h, size_matY, hipMemcpyHostToDevice);

  hipMemcpy(matw1_d, matw1_h, size_matw1, hipMemcpyHostToDevice);
  hipMemcpy(matw2_d, matw2_h, size_matw2, hipMemcpyHostToDevice);
  hipMemcpy(matw3_d, matw3_h, size_matw3, hipMemcpyHostToDevice);

  hipMemcpy(vecb1_d, vecb1_h, size_vecb1, hipMemcpyHostToDevice);
  hipMemcpy(vecb2_d, vecb2_h, size_vecb2, hipMemcpyHostToDevice);
  hipMemcpy(vecb3_d, vecb3_h, size_vecb3, hipMemcpyHostToDevice);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////computation///////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////  

  // forward propagation
  float lr = 0.1;
  compute_nn(lr, matw1_d, matw2_d, matw3_d, vecb1_d, vecb2_d, vecb3_d, matX_d, matY_d, Ns, N0, N1, N2, N3);

  // Retrieve result from device and store it in host array
  hipMemcpy(matw1_h, matw1_d, size_matw1, hipMemcpyDeviceToHost);
  hipMemcpy(matw2_h, matw2_d, size_matw2, hipMemcpyDeviceToHost);
  hipMemcpy(matw3_h, matw3_d, size_matw3, hipMemcpyDeviceToHost);

  hipMemcpy(vecb1_h, vecb1_d, size_vecb1, hipMemcpyDeviceToHost);
  hipMemcpy(vecb2_h, vecb2_d, size_vecb2, hipMemcpyDeviceToHost);
  hipMemcpy(vecb3_h, vecb3_d, size_vecb3, hipMemcpyDeviceToHost);

  // Print results
  print_matrix(matw3_h, N3, N2);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////finalization//////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////    
  
  // Cleanup
  delete [] matX_h;
  delete [] matY_h;
  delete [] matw1_h;
  delete [] matw2_h;
  delete [] matw3_h;
  delete [] vecb1_h;
  delete [] vecb2_h;
  delete [] vecb3_h;

  hipFree(matX_d);
  hipFree(matY_d);
  hipFree(matw1_d);
  hipFree(matw2_d);
  hipFree(matw3_d);
  hipFree(vecb1_d);
  hipFree(vecb2_d);
  hipFree(vecb3_d);

  return 0;
}