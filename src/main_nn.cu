#include "hip/hip_runtime.h"
// main.cpp : Defines the entry point for the console application.
//
#include <iostream>
#include <math.h>
#include "hipblas.h"
#include "util.h"
#include "nn.h"
#include "read.h"
#include "test.h"

// main routine that executes on the host
int main(void){

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  /////////////////////////declaration and allocation///////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////

  // matrix and vector size
  const int Ns = 60000;   // matB row/matA column number
  const int N0 = 784;  // matC row/matA row number
  const int N1 = 256;  // matC column/matB column number
  const int N2 = 128;  // matB row/matA column number
  const int N3 = 10;   // matB row/matA column number

  const int Ntrain = 60000;
  const int Ntest  = 10000;

  float lr = 0.1; // learning rate
  int epoch = 500;  // max iteration

  // allocate on host

  // data X, Y
  int *x_train   = new int[N0 * Ntrain];
  int *x_test    = new int[N0 * Ntest];
  int *y_train   = new int[Ntrain];
  int *y_test    = new int[Ntest];

  float *matX_h  = new float[N0 * Ns];
  float *matY_h  = new float[N3 * Ns];

  // weights
  float *matw1_h  = new float[N1 * N0];
  float *matw2_h  = new float[N2 * N1];
  float *matw3_h  = new float[N3 * N2];
  
  // biases
  float *vecb1_h  = new float[N1];
  float *vecb2_h  = new float[N2];
  float *vecb3_h  = new float[N3];
  
  // Pointer to device arrays
  float *matX_d, *matY_d;
  float *matw1_d, *matw2_d, *matw3_d;
  float *vecb1_d, *vecb2_d, *vecb3_d;

  // size of matrix
  size_t size_matX  = N0 * Ns * sizeof(float);
  size_t size_matY  = N3 * Ns * sizeof(float);

  size_t size_matw1 = N1 * N0 * sizeof(float);
  size_t size_matw2 = N2 * N1 * sizeof(float);
  size_t size_matw3 = N3 * N2 * sizeof(float);

  // size of vector
  size_t size_vecb1 = N1 * sizeof(float);
  size_t size_vecb2 = N2 * sizeof(float);
  size_t size_vecb3 = N3 * sizeof(float);

  // allocate on device
  hipMalloc((void **) &matX_d, size_matX);
  hipMalloc((void **) &matY_d, size_matY);

  hipMalloc((void **) &matw1_d, size_matw1);
  hipMalloc((void **) &matw2_d, size_matw2);
  hipMalloc((void **) &matw3_d, size_matw3);

  hipMalloc((void **) &vecb1_d, size_vecb1);
  hipMalloc((void **) &vecb2_d, size_vecb2);
  hipMalloc((void **) &vecb3_d, size_vecb3);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////initialization////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////

  printf("initialization start\n");

  // print matrix size
  std::cout << " X matrix dims = (" << N0 << ", " << Ns << ")" << std::endl;
  std::cout << " Y matrix dims = (" << N0 << ", " << Ns << ")" << std::endl;

  std::cout << " w1 matrix dims = (" << N1 << ", " << N0 << ")" << std::endl;
  std::cout << " w2 matrix dims = (" << N2 << ", " << N1 << ")" << std::endl;
  std::cout << " w3 matrix dims = (" << N3 << ", " << N2 << ")" << std::endl;

  std::cout << " b1 vector dims = (" << N1 << ")" << std::endl;
  std::cout << " b2 vector dims = (" << N2 << ")" << std::endl;
  std::cout << " b3 vector dims = (" << N3 << ")" << std::endl;

  // Initialize host array and copy it to CUDA device
  // random_matrix(matX_h, N0, Ns, 0.0f);
  // random_matrix(matY_h, N3, Ns, 0.0f);

  // read mnist data
  read_mnist(matX_h, matY_h, x_train, x_test, y_test, y_train, N0, N3, Ntrain, Ntest);

  // print data
  int idx = 41576;
  for (int i=0; i<N0; i++) {
    printf("%d ", (int)(matX_h[idx * N0 + i] > 0.0f));
      if ((i+1) % 28 == 0) putchar('\n');
  }
  int idx_Y = std::distance(matY_h  + idx * N3, std::max_element(matY_h + idx * N3, matY_h  + (idx + 1) * N3 - 1));
  printf("y=%d, %d\n", y_train[idx], idx_Y);

  random_matrix(matw1_h, N1, N0, 0.5f);
  random_matrix(matw2_h, N2, N1, 0.5f);
  random_matrix(matw3_h, N3, N2, 0.5f);

  random_vector(vecb1_h, N1);
  random_vector(vecb2_h, N2);
  random_vector(vecb3_h, N3);
  
  // copy data from host to device
  hipMemcpy(matX_d, matX_h, size_matX, hipMemcpyHostToDevice);
  hipMemcpy(matY_d, matY_h, size_matY, hipMemcpyHostToDevice);

  hipMemcpy(matw1_d, matw1_h, size_matw1, hipMemcpyHostToDevice);
  hipMemcpy(matw2_d, matw2_h, size_matw2, hipMemcpyHostToDevice);
  hipMemcpy(matw3_d, matw3_h, size_matw3, hipMemcpyHostToDevice);

  hipMemcpy(vecb1_d, vecb1_h, size_vecb1, hipMemcpyHostToDevice);
  hipMemcpy(vecb2_d, vecb2_h, size_vecb2, hipMemcpyHostToDevice);
  hipMemcpy(vecb3_d, vecb3_h, size_vecb3, hipMemcpyHostToDevice);

  printf("initialization end\n");

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////computation///////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////

  // forward propagation
  printf("computation start\n");
  // printf("b3= "); print_vector(vecb3_h, N3);
  compute_nn(lr, epoch, matw1_d, matw2_d, matw3_d, vecb1_d, vecb2_d, vecb3_d, matX_d, matY_d, matY_h, Ns, N0, N1, N2, N3);

  // Retrieve result from device and store it in host array
  hipMemcpy(matw1_h, matw1_d, size_matw1, hipMemcpyDeviceToHost);
  hipMemcpy(matw2_h, matw2_d, size_matw2, hipMemcpyDeviceToHost);
  hipMemcpy(matw3_h, matw3_d, size_matw3, hipMemcpyDeviceToHost);

  hipMemcpy(vecb1_h, vecb1_d, size_vecb1, hipMemcpyDeviceToHost);
  hipMemcpy(vecb2_h, vecb2_d, size_vecb2, hipMemcpyDeviceToHost);
  hipMemcpy(vecb3_h, vecb3_d, size_vecb3, hipMemcpyDeviceToHost);

  // Print results
  // printf("b3= "); print_vector(vecb3_h, N3);
  // print_matrix(matw3_h, N3, N2);

  printf("computation end\n");

  //////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////finalization//////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////////////////////////////    
  
  // Cleanup
  delete [] matX_h;
  delete [] matY_h;
  delete [] matw1_h;
  delete [] matw2_h;
  delete [] matw3_h;
  delete [] vecb1_h;
  delete [] vecb2_h;
  delete [] vecb3_h;
  delete [] x_train;
  delete [] x_test;
  delete [] y_train;
  delete [] y_test;

  hipFree(matX_d);
  hipFree(matY_d);
  hipFree(matw1_d);
  hipFree(matw2_d);
  hipFree(matw3_d);
  hipFree(vecb1_d);
  hipFree(vecb2_d);
  hipFree(vecb3_d);

  return 0;
}